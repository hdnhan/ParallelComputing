#include "hip/hip_runtime.h"
#include <hipblas.h>     // hipblasHandle_t, hipblasCreate, hipblasDestroy, hipblasSaxpy, hipblasDaxpy
#include <hip/hip_runtime.h>  // hipMalloc, hipMemcpy, hipFree

#include <cstdint>    // uint64_t
#include <stdexcept>  // std::runtime_error

template <typename T>
__global__ void cuda_gemm_kernel(T const alpha, T const *dA, T const *dB, T const beta, T *dC,
                                 uint64_t const rows, uint64_t const N,  uint64_t const cols) {
    uint64_t rid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t cid = blockIdx.y * blockDim.y + threadIdx.y;

    if (rid >= rows || cid >= cols) return;
    T tmp = 0;
    for (int k = 0; k < N; k++) tmp += dA[rid * N + k] * dB[k * cols + cid];
    dC[rid * cols + cid] = alpha * tmp + beta * dC[rid * cols + cid];
}

// CUDA matrix-matrix multiplication
template <typename T>
void cuda_gemm(T const alpha, T const *hA, T const *hB, T const beta, T *hC, uint64_t const rows,
               uint64_t const N, uint64_t const cols) {
    // Allocate device memory
    T *dA, *dB, *dC;
    hipMalloc(&dA, rows * N * sizeof(T));
    hipMalloc(&dB, N * cols * sizeof(T));
    hipMalloc(&dC, rows * cols * sizeof(T));

    // Copy data to the device
    hipMemcpy(dA, hA, rows * N * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, N * cols * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(dC, hC, rows * cols * sizeof(T), hipMemcpyHostToDevice);

    // CUDA kernel
    // Threads per CTA dimension
    uint64_t const RTHREADS = 32;
    uint64_t const CTHREADS = 32;

    // Blocks per grid dimension
    uint64_t const RBLOCKS = (max(rows, N) + RTHREADS - 1) / RTHREADS;
    uint64_t const CBLOCKS = (max(N, cols) + CTHREADS - 1) / CTHREADS;

    // Use dim3 structs for block  and grid dimensions
    dim3 threads(RTHREADS, CTHREADS);
    dim3 blocks(RBLOCKS, CBLOCKS);
    cuda_gemm_kernel<<<blocks, threads>>>(alpha, dA, dB, beta, dC, rows, N, cols);
    hipMemcpy(hC, dC, rows * cols * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}

// cuBLAS matrix-matrix multiplication
template <typename T>
void cublas_gemm(T const alpha, T const *hA, T const *hB, T const beta, T *hC, uint64_t const rows,
                 uint64_t const N, uint64_t const cols) {
    // Allocate device memory
    double *dA, *dB, *dC;
    hipMalloc(&dA, rows * N * sizeof(double));
    hipMalloc(&dB, N * cols * sizeof(double));
    hipMalloc(&dC, rows * cols * sizeof(double));

    // Copy data to the device
    hipMemcpy(dA, hA, rows * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, N * cols * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dC, hC, rows * cols * sizeof(double), hipMemcpyHostToDevice);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // cuBLAS gemm
    // Calculate: C = (alpha * A) * B + (beta * C)
    // (rows x N) * (N x cols) = (rows x cols)
    // See A and B as row-major matrices and outputs a row-major matrix dC
    if constexpr (std::is_same<T, float>::value)
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cols, rows, N, &alpha, dB, cols, dA, N, &beta, dC,
                    cols);
    else if constexpr (std::is_same<T, double>::value)
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cols, rows, N, &alpha, dB, cols, dA, N, &beta, dC,
                    cols);
    else
        throw std::runtime_error("cublas_gemm: unsupported type");
    hipMemcpy(hC, dC, rows * cols * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);
}
