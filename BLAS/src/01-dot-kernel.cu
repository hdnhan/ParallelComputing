#include <hipblas.h>     // cublasHandle_t, cublasCreate_v2, cublasDestroy_v2, cublasSaxpy, cublasDaxpy
#include <hip/hip_runtime.h>  // cudaMalloc, cudaMemcpy, cudaFree

#include <cstdint>    // uint64_t
#include <stdexcept>  // std::runtime_error

template <typename T>
T cublas_dot(T const *ha, T const *hb, uint64_t const N) {
    // Allocate memory on the device-side (GPU-side)
    T *da, *db;

    hipMalloc(&da, N * sizeof(T));
    hipMalloc(&db, N * sizeof(T));

    // Copy data from the host to the device (CPU -> GPU)
    hipMemcpy(da, ha, N * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, N * sizeof(T), hipMemcpyHostToDevice);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform the dot product
    T res = 0;
    if constexpr (std::is_same<T, float>::value)
        hipblasSdot(handle, N, da, 1, db, 1, &res);
    else if constexpr (std::is_same<T, double>::value)
        hipblasDdot(handle, N, da, 1, db, 1, &res);
    else
        throw std::runtime_error("Unsupported type");

    // Free the device memory
    hipFree(da);
    hipFree(db);
    hipblasDestroy(handle);
    return res;
}